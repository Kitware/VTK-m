#include "hip/hip_runtime.h"
//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <vtkm/testing/Testing.h>

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include <vtkm/exec/FunctorBase.h>
#include <vtkm/exec/arg/BasicArg.h>
#include <vtkm/exec/arg/ThreadIndicesBasic.h>
#include <vtkm/exec/cuda/internal/TaskStrided.h>

#include <vtkm/StaticAssert.h>

#include <vtkm/internal/FunctionInterface.h>
#include <vtkm/internal/Invocation.h>

#if defined(VTKM_MSVC)
#pragma warning(push)
#pragma warning(disable : 4068) //unknown pragma
#endif

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
// Disable warning "declared but never referenced"
// This file produces several false-positive warnings
// Eg: TestExecObject::TestExecObject, MyOutputToInputMapPortal::Get,
//     TestWorkletProxy::operator()
#pragma push
#if (CUDART_VERSION >= 11050)
#pragma nv_diag_suppress 177
#else
#pragma diag_suppress 177
#endif

#endif

namespace
{

struct TestExecObject
{
  using PortalType = vtkm::cont::ArrayHandle<vtkm::Id>::WritePortalType;

  VTKM_EXEC_CONT
  TestExecObject(PortalType portal)
    : Portal(portal)
  {
  }

  PortalType Portal;
};

struct MyOutputToInputMapPortal
{
  using ValueType = vtkm::Id;
  VTKM_EXEC_CONT
  vtkm::Id Get(vtkm::Id index) const { return index; }
};

struct MyVisitArrayPortal
{
  using ValueType = vtkm::IdComponent;
  VTKM_EXEC_CONT
  vtkm::IdComponent Get(vtkm::Id) const { return 1; }
};

struct MyThreadToOutputMapPortal
{
  using ValueType = vtkm::Id;
  VTKM_EXEC_CONT
  vtkm::Id Get(vtkm::Id index) const { return index; }
};

struct TestFetchTagInput
{
};
struct TestFetchTagOutput
{
};

// Missing TransportTag, but we are not testing that so we can leave it out.
struct TestControlSignatureTagInput
{
  using FetchTag = TestFetchTagInput;
};
struct TestControlSignatureTagOutput
{
  using FetchTag = TestFetchTagOutput;
};

} // anonymous namespace

namespace vtkm
{
namespace exec
{
namespace arg
{

template <>
struct Fetch<TestFetchTagInput, vtkm::exec::arg::AspectTagDefault, TestExecObject>
{
  using ValueType = vtkm::Id;

  VTKM_EXEC
  ValueType Load(const vtkm::exec::arg::ThreadIndicesBasic& indices,
                 const TestExecObject& execObject) const
  {
    return execObject.Portal.Get(indices.GetInputIndex()) + 10 * indices.GetInputIndex();
  }

  VTKM_EXEC
  void Store(const vtkm::exec::arg::ThreadIndicesBasic&, const TestExecObject&, ValueType) const
  {
    // No-op
  }
};

template <>
struct Fetch<TestFetchTagOutput, vtkm::exec::arg::AspectTagDefault, TestExecObject>
{
  using ValueType = vtkm::Id;

  VTKM_EXEC
  ValueType Load(const vtkm::exec::arg::ThreadIndicesBasic&, const TestExecObject&) const
  {
    // No-op
    return ValueType();
  }

  VTKM_EXEC
  void Store(const vtkm::exec::arg::ThreadIndicesBasic& indices,
             const TestExecObject& execObject,
             ValueType value) const
  {
    execObject.Portal.Set(indices.GetOutputIndex(), value + 20 * indices.GetOutputIndex());
  }
};
}
}
} // vtkm::exec::arg

namespace
{

using TestControlSignature = void(TestControlSignatureTagInput, TestControlSignatureTagOutput);
using TestControlInterface = vtkm::internal::FunctionInterface<TestControlSignature>;

using TestExecutionSignature1 = void(vtkm::exec::arg::BasicArg<1>, vtkm::exec::arg::BasicArg<2>);
using TestExecutionInterface1 = vtkm::internal::FunctionInterface<TestExecutionSignature1>;

using TestExecutionSignature2 = vtkm::exec::arg::BasicArg<2>(vtkm::exec::arg::BasicArg<1>);
using TestExecutionInterface2 = vtkm::internal::FunctionInterface<TestExecutionSignature2>;

using ExecutionParameterInterface =
  vtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)>;

using InvocationType1 = vtkm::internal::Invocation<ExecutionParameterInterface,
                                                   TestControlInterface,
                                                   TestExecutionInterface1,
                                                   1,
                                                   MyOutputToInputMapPortal,
                                                   MyVisitArrayPortal,
                                                   MyThreadToOutputMapPortal>;

using InvocationType2 = vtkm::internal::Invocation<ExecutionParameterInterface,
                                                   TestControlInterface,
                                                   TestExecutionInterface2,
                                                   1,
                                                   MyOutputToInputMapPortal,
                                                   MyVisitArrayPortal,
                                                   MyThreadToOutputMapPortal>;

template <typename TaskType>
static __global__ void ScheduleTaskStrided(TaskType task, vtkm::Id start, vtkm::Id end)
{

  const vtkm::Id index = blockIdx.x * blockDim.x + threadIdx.x;
  const vtkm::Id inc = blockDim.x * gridDim.x;
  if (index >= start && index < end)
  {
    task(index, end, inc);
  }
}

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletProxy : vtkm::exec::FunctorBase
{
  VTKM_EXEC
  void operator()(vtkm::Id input, vtkm::Id& output) const { output = input + 100; }

  VTKM_EXEC
  vtkm::Id operator()(vtkm::Id input) const { return input + 200; }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType>
  VTKM_EXEC vtkm::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&) const
  {
    vtkm::Id outIndex = threadToOut.Get(threadIndex);
    return vtkm::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex);
  }
};

#define ERROR_MESSAGE "Expected worklet error."

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletErrorProxy : vtkm::exec::FunctorBase
{
  VTKM_EXEC
  void operator()(vtkm::Id, vtkm::Id) const { this->RaiseError(ERROR_MESSAGE); }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType>
  VTKM_EXEC vtkm::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&) const
  {
    vtkm::Id outIndex = threadToOut.Get(threadIndex);
    return vtkm::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex);
  }
};

// Check behavior of InvocationToFetch helper class.

VTKM_STATIC_ASSERT(
  (std::is_same<
    vtkm::exec::internal::detail::
      InvocationToFetch<vtkm::exec::arg::ThreadIndicesBasic, InvocationType1, 1>::type,
    vtkm::exec::arg::Fetch<TestFetchTagInput, vtkm::exec::arg::AspectTagDefault, TestExecObject>>::
     type::value));

VTKM_STATIC_ASSERT(
  (std::is_same<
    vtkm::exec::internal::detail::
      InvocationToFetch<vtkm::exec::arg::ThreadIndicesBasic, InvocationType1, 2>::type,
    vtkm::exec::arg::Fetch<TestFetchTagOutput, vtkm::exec::arg::AspectTagDefault, TestExecObject>>::
     type::value));

VTKM_STATIC_ASSERT(
  (std::is_same<
    vtkm::exec::internal::detail::
      InvocationToFetch<vtkm::exec::arg::ThreadIndicesBasic, InvocationType2, 0>::type,
    vtkm::exec::arg::Fetch<TestFetchTagOutput, vtkm::exec::arg::AspectTagDefault, TestExecObject>>::
     type::value));

template <typename DeviceAdapter>
void TestNormalFunctorInvoke()
{
  std::cout << "Testing normal worklet invoke." << std::endl;

  vtkm::cont::Token token;

  vtkm::Id inputTestValues[3] = { 5, 5, 6 };

  vtkm::cont::ArrayHandle<vtkm::Id> input =
    vtkm::cont::make_ArrayHandle(inputTestValues, 3, vtkm::CopyFlag::Off);
  vtkm::cont::ArrayHandle<vtkm::Id> output;

  vtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    vtkm::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
      TestExecObject(output.PrepareForOutput(3, DeviceAdapter(), token)));

  std::cout << "  Try void return." << std::endl;
  TestWorkletProxy worklet;
  InvocationType1 invocation1(execObjects);

  using TaskTypes = typename vtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task1 = TaskTypes::MakeTask(worklet, invocation1, vtkm::Id());

  ScheduleTaskStrided<decltype(task1)><<<32, 256>>>(task1, 1, 2);
  hipDeviceSynchronize();
  token.DetachFromAll();
  input.SyncControlArray();
  output.SyncControlArray();

  VTKM_TEST_ASSERT(inputTestValues[1] == 5, "Input value changed.");
  VTKM_TEST_ASSERT(output.ReadPortal().Get(1) == inputTestValues[1] + 100 + 30,
                   "Output value not set right.");

  std::cout << "  Try return value." << std::endl;

  execObjects = vtkm::internal::make_FunctionInterface<void>(
    TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
    TestExecObject(output.PrepareForOutput(3, DeviceAdapter(), token)));

  InvocationType2 invocation2(execObjects);

  using TaskTypes = typename vtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task2 = TaskTypes::MakeTask(worklet, invocation2, vtkm::Id());

  ScheduleTaskStrided<decltype(task2)><<<32, 256>>>(task2, 2, 3);
  hipDeviceSynchronize();
  token.DetachFromAll();
  input.SyncControlArray();
  output.SyncControlArray();

  VTKM_TEST_ASSERT(inputTestValues[2] == 6, "Input value changed.");
  VTKM_TEST_ASSERT(output.ReadPortal().Get(2) == inputTestValues[2] + 200 + 30 * 2,
                   "Output value not set right.");
}

template <typename DeviceAdapter>
void TestErrorFunctorInvoke()
{
  std::cout << "Testing invoke with an error raised in the worklet." << std::endl;

  vtkm::cont::Token token;

  vtkm::Id inputTestValue = 5;
  vtkm::Id outputTestValue = static_cast<vtkm::Id>(0xDEADDEAD);

  vtkm::cont::ArrayHandle<vtkm::Id> input =
    vtkm::cont::make_ArrayHandle(&inputTestValue, 1, vtkm::CopyFlag::Off);
  vtkm::cont::ArrayHandle<vtkm::Id> output =
    vtkm::cont::make_ArrayHandle(&outputTestValue, 1, vtkm::CopyFlag::Off);

  vtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    vtkm::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
      TestExecObject(output.PrepareForInPlace(DeviceAdapter(), token)));

  using TaskStrided1 = vtkm::exec::cuda::internal::
    TaskStrided1D<TestWorkletErrorProxy, InvocationType1, vtkm::cont::internal::HintList<>>;
  TestWorkletErrorProxy worklet;
  InvocationType1 invocation(execObjects);

  using TaskTypes = typename vtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  using Algorithm = vtkm::cont::DeviceAdapterAlgorithm<DeviceAdapter>;

  auto task = TaskTypes::MakeTask(worklet, invocation, vtkm::Id());

  auto errorArray = Algorithm::GetPinnedErrorArray();
  vtkm::exec::internal::ErrorMessageBuffer errorMessage(errorArray.DevicePtr, errorArray.Size);
  task.SetErrorMessageBuffer(errorMessage);

  ScheduleTaskStrided<decltype(task)><<<32, 256>>>(task, 1, 2);
  hipDeviceSynchronize();

  VTKM_TEST_ASSERT(errorMessage.IsErrorRaised(), "Error not raised correctly.");
  VTKM_TEST_ASSERT(errorArray.HostPtr == std::string(ERROR_MESSAGE), "Got wrong error message.");
}

template <typename DeviceAdapter>
void TestTaskStrided()
{
  TestNormalFunctorInvoke<DeviceAdapter>();
  TestErrorFunctorInvoke<DeviceAdapter>();
}

} // anonymous namespace

int UnitTestTaskStrided(int argc, char* argv[])
{
  return vtkm::testing::Testing::Run(TestTaskStrided<vtkm::cont::DeviceAdapterTagCuda>, argc, argv);
}

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
#pragma pop
#endif

#if defined(VTKM_MSVC)
#pragma warning(pop)
#endif
